
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>

/*
User includes the size of the square A and B in terminal line
*/

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
 
    // Destroy the handle
    hipblasDestroy(handle);
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
    for(int i = 0; i < nr_rows_A; ++i){ 
        for(int j = 0; j < nr_cols_A; ++j){
           std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main(int argc, char *argv[]){

    if(argc != 2){
        std::cout << "The usage is ./bmark 'sizeof matrix' " << std::endl;
    }

    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = atoi(argv[1]);

    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    for(int i = 0; i<nr_rows_A; i++){
        for(int j = 0; j<nr_cols_A; j++){
            h_A[(j*nr_rows_A) + (i)] = (i*nr_rows_A) + (j+1);
            // std::cout << h_A[(i*nr_rows_A) + (j+1)] << " ";
            // if(j == nr_cols_A-1){
            //     std::cout << std::endl;
            // }
            h_B[(j*nr_rows_B) + (i)] = (i*nr_rows_A) + (j+1);
        }
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice);


    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
    // Copy (and print) the result on host memory
    hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "C =" << std::endl;
    print_matrix(h_C, nr_rows_C, nr_cols_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
    free(h_A);
    free(h_B);
    free(h_C);
 
    return 0;
}