#include <cstdlib>
#include <hipblas.h>

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
 
    // Destroy the handle
    hipblasDestroy(handle);
}

void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
    for(int i = 0; i < nr_rows_A; ++i){ 
        for(int j = 0; j < nr_cols_A; ++j){
           std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main(){

    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_rows_c = 3;

    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    for(int i = 0; i<nr_rows_A; i++){
        for(int j = 0; j<nr_cols_A; j++){
            h_A[i][j] = (i*nr_rows_A) + (j+1);
            h_B[i][j] = (i*nr_rows_A) + (j+1);
        }
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
    free(h_A);
    free(h_B);
    free(h_C);
 
    return 0;
}