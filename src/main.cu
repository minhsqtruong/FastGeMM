#include "hip/hip_runtime.h"
#include "fastgemm.cuh"
using namespace std;
int main(int argc, char const *argv[]) {

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int K = atoi(argv[3]);

  #ifdef DEBUG
  struct hipDeviceProp_t prop;
  int device = 0;
  hipGetDeviceProperties(&prop, device);
  cout << "Device name: " << prop.name << endl;
  cout << "Total Global Memory (bytes): " << prop.totalGlobalMem << endl;
  cout << "Shared Memory per Block (bytes): " << prop.sharedMemPerBlock << endl;
  cout << "L1 Cache Size (bytes): 16000" << endl;
  cout << "L2 Cache Size (bytes): " << prop.l2CacheSize << endl;
  cout << "Cache Line Size (bytes): 128" << endl;
  cout << "Registers per Block: " << prop.regsPerBlock << endl;
  cout << "Warp Size: " << prop.warpSize << endl;
  cout << "Number of Warp Schedulers: 4" << endl;
  cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
  cout << "Compute Capability: " << prop.major << endl;
  cout << "Clock Rate (kHz): " << prop.clockRate << endl;
  cout << "Number of SM: " << prop.multiProcessorCount << endl;
  #endif

  // 1) Run reference code <DEBUG means run dumb CPU, REAL means CUBLAS>
  #ifdef DEBUG

  cout << "(M,N,K) = " << M << " " << N << " " << K << endl;

  float* C = (float*) malloc(sizeof(float) * M * N);
  float* A = (float*) malloc(sizeof(float) * M * K);
  float* B = (float*) malloc(sizeof(float) * K * N);

  for (int i = 0; i < M * N; i++)
    C[i] = 0.0;
  for (int i = 0; i < M * K; i++)
    A[i] = (float) i;
  for (int i = 0; i < M * N; i++)
    B[i] = (float) i;

  ref_mmul(C, A, B, M, N, K);

  cout << "Reference A: " << endl;
  printMatrix(A,M,K);
  cout << "Reference B: " << endl;
  printMatrix(B,K,N);
  cout << "Reference C: " << endl;
  printMatrix(C,M,N);

  // 2) Pack Data for Kernel <THIS CHANGE AS THE CODE PROGRESSES>
  cout << "Start initializing device arrays" << endl;
  float4* C_gpu;
  float4* A_gpu;
  float4* B_gpu;
  cout << hipMallocManaged(&C_gpu, sizeof(float4) * (M * N)/4) << endl;;
  hipMallocManaged(&A_gpu, sizeof(float4) * (M * K)/4);
  hipMallocManaged(&B_gpu, sizeof(float4) * (K * N)/4);

  cout << "Start loading device arrays" << endl;
  for (int i = 0; i < (M * K)/4; i++)
    A_gpu[i] = make_float4(A[i*4 + 0],A[i*4 + 1],A[i*4 + 2],A[i*4 + 3]);
  for (int i = 0; i < (K * N)/4; i++)
    B_gpu[i] = make_float4(B[i*4 + 0],B[i*4 + 1],B[i*4 + 2],B[i*4 + 3]);

  cout << "Start fastgemm" << endl;
  launchFastGemm(C_gpu, A_gpu, B_gpu, M, N, K);

  free(A);
  free(B);
  free(C);
  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(C_gpu);
  #endif

  #ifdef REAL
  // INSERT CUBLAS HERE
  #endif

  return 0;
}

