#include "fastgemm.cuh"
using namespace std;
int main(int argc, char const *argv[]) {

  int M = atoi(argv[1]);
  int N = atoi(argv[2]);
  int K = atoi(argv[3]);

  #ifdef DEBUG
  struct hipDeviceProp_t prop;
  int device = 0;
  hipGetDeviceProperties(&prop, device);
  cout << "Device name: " << prop.name << endl;
  cout << "Total Global Memory (bytes): " << prop.totalGlobalMem << endl;
  cout << "Shared Memory per Block (bytes): " << prop.sharedMemPerBlock << endl;
  cout << "L1 Cache Size (bytes): 16000" << endl;
  cout << "L2 Cache Size (bytes): " << prop.l2CacheSize << endl;
  cout << "Cache Line Size (bytes): 128" << endl;
  cout << "Registers per Block: " << prop.regsPerBlock << endl;
  cout << "Warp Size: " << prop.warpSize << endl;
  cout << "Number of Warp Schedulers: 4" << endl;
  cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
  cout << "Compute Capability: " << prop.major << endl;
  cout << "Clock Rate (kHz): " << prop.clockRate << endl;
  cout << "Number of SM: " << prop.multiProcessorCount << endl;
  #endif

  // 1) Run reference code <DEBUG means run dumb CPU, REAL means CUBLAS>
  #ifdef DEBUG
  float* C = (float*) malloc(sizeof(float) * M * N);
  float* A = (float*) malloc(sizeof(float) * M * K);
  float* B = (float*) malloc(sizeof(float) * K * N);

  for (int i = 0; i < M * N; i++)
    C[i] = 0.0;
  for (int i = 0; i < M * K; i++)
    A[i] = (float) i;
  for (int i = 0; i < M * N; i++)
    B[i] = (float) i;

  ref_mmul(C, A, B, M, N, K);

  cout << "Reference A: " << endl;
  printMatrix(A,M,K);
  cout << "Reference B: " << endl;
  printMatrix(B,K,N);
  cout << "Reference C: " << endl;
  printMatrix(C,M,N);
  #endif

  #ifdef REAL
  // INSERT CUBLAS HERE
  #endif
  return 0;
}

